
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel (int a, int b, int* c) {
	*c = a * b;
}

int main (void) {
	int c;
	int *dev_c;

	hipMalloc(&dev_c, sizeof(int));
	kernel<<<1,1>>>(2, 3, dev_c);

	hipMemcpy(&c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	printf("%d x %d = %d\n", 2, 3, c);
	hipFree(dev_c);

	return 0;
}